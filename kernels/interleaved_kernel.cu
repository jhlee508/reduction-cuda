
#include <hip/hip_runtime.h>
__global__ void interleaved_kernel(double* arr, int size, double* res) {
  extern __shared__ double s_arr[];

  int lid = threadIdx.x;
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  /* 1. Load to SMEM */
  s_arr[lid] = (gid < size) ? arr[gid] : 0;
  __syncthreads(); 

  /* 2. Reduction in SMEM */
  for (int s = 1; s < blockDim.x; s *= 2) { // s = 1, 2, ..., 512
    if (lid % (2 * s) == 0) { // lid: (0, 2, ..., 1022), (0, 4, ..., 1020), ..., (0, 512)
      s_arr[lid] += s_arr[lid + s];
    }
    __syncthreads();
  }

  /* 3. Store to GMEM */
  if (lid == 0) { res[blockIdx.x] = s_arr[0]; }
}