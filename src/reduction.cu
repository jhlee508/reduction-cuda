#include <cstdio>
#include <hipblas.h>

#include "reduction.h"
#include "kernels.cuh"

static double* d_arr;
static double* d_output;

static hipblasHandle_t handle;
static double* d_cublas_output;
static double* d_ones;

double cpu_single(double* arr, int size) {
  double sum = 0.0f;
  for (int i = 0; i < size; i++) { 
    sum += arr[i]; 
  }
  return sum;
}

double cpu_multithreading(double* arr, int size) {
  double sum = 0.0f;
#pragma omp parallel for reduction(+:sum)
  for (int i = 0; i < size; i++) { 
    sum += arr[i]; 
  }
  return sum;
}

double reduction_cpu(double* arr, int size) {
  double sum = 0.0;
  
  /* CPU Single Core */
  // sum = cpu_single(arr, size);

  /* CPU Multi-threading */
  // sum = cpu_multithreading(arr, size);

  return sum;
}

void reduction(double* arr, int size) {

  /* Total reduction is computed in 4 steps or levels (kernels) 
     with BLOCK_SIZE 256 and 33554432 elements as follows:
      - Level 0: CEIL_DIV(33554432, 256) = 131072 blocks
      - Level 1: CEIL_DIV(131072, 256) = 512 blocks
      - Level 2: CEIL_DIV(512, 256) = 2 blocks
      - Level 3: CEIL_DIV(2, 256) = 1 blocks
  */

  /* 1. Interleaved Addressing */
  // dim3 gridDim0(CEIL_DIV(size, BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, BLOCK_SIZE));
  // dim3 gridDim3(CEIL_DIV(gridDim2.x, BLOCK_SIZE));
  // interleaved_kernel<<<gridDim0, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // interleaved_kernel<<<gridDim1, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // interleaved_kernel<<<gridDim2, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  // interleaved_kernel<<<gridDim3, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim2.x, d_output);

  /* 2. Interleaved Addressing (Contiguous Threads) */
  // dim3 gridDim0(CEIL_DIV(size, BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, BLOCK_SIZE));
  // dim3 gridDim3(CEIL_DIV(gridDim2.x, BLOCK_SIZE));
  // interleaved_contiguous_kernel<<<gridDim0, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // interleaved_contiguous_kernel<<<gridDim1, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // interleaved_contiguous_kernel<<<gridDim2, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  // interleaved_contiguous_kernel<<<gridDim3, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim2.x, d_output);

  /* 3. Sequential Addressing */
  // dim3 gridDim0(CEIL_DIV(size, BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, BLOCK_SIZE));
  // dim3 gridDim3(CEIL_DIV(gridDim2.x, BLOCK_SIZE));
  // sequential_kernel<<<gridDim0, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // sequential_kernel<<<gridDim1, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // sequential_kernel<<<gridDim2, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  // sequential_kernel<<<gridDim3, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim2.x, d_output);

  /* 4. Sequential Addressing (Multiple Load per Thread) */
  // const int HALF_BLOCK_SIZE = BLOCK_SIZE / 2;
  // dim3 gridDim0(CEIL_DIV(size, BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, BLOCK_SIZE));
  // dim3 gridDim3(CEIL_DIV(gridDim2.x, BLOCK_SIZE));
  // sequential_multi_load_kernel<<<gridDim0, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // sequential_multi_load_kernel<<<gridDim1, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // sequential_multi_load_kernel<<<gridDim2, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  // sequential_multi_load_kernel<<<gridDim3, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim2.x, d_output);

  /* 5. Sequential Addressing (Warp Shuffle Last Warp) */
  // const int HALF_BLOCK_SIZE = BLOCK_SIZE / 2;
  // dim3 gridDim0(CEIL_DIV(size, BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, BLOCK_SIZE));
  // dim3 gridDim3(CEIL_DIV(gridDim2.x, BLOCK_SIZE));
  // sequential_warp_shfl_last_kernel<<<gridDim0, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // sequential_warp_shfl_last_kernel<<<gridDim1, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // sequential_warp_shfl_last_kernel<<<gridDim2, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  // sequential_warp_shfl_last_kernel<<<gridDim3, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim2.x, d_output);
  
  /* 6. Sequential Addressing (Unroll Last Warp) */
  // const int HALF_BLOCK_SIZE = BLOCK_SIZE / 2;
  // dim3 gridDim0(CEIL_DIV(size, BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, BLOCK_SIZE));
  // dim3 gridDim3(CEIL_DIV(gridDim2.x, BLOCK_SIZE));
  // sequential_unroll_last_kernel<<<gridDim0, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // sequential_unroll_last_kernel<<<gridDim1, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // sequential_unroll_last_kernel<<<gridDim2, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  // sequential_unroll_last_kernel<<<gridDim3, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim2.x, d_output);

  /* 7. Sequential Addressing (Unroll All) */
  // const int HALF_BLOCK_SIZE = BLOCK_SIZE / 2;
  // dim3 gridDim0(CEIL_DIV(size, BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, BLOCK_SIZE));
  // dim3 gridDim3(CEIL_DIV(gridDim2.x, BLOCK_SIZE));
  // sequential_unroll_all_kernel<<<gridDim0, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // sequential_unroll_all_kernel<<<gridDim1, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // sequential_unroll_all_kernel<<<gridDim2, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  // sequential_unroll_all_kernel<<<gridDim3, HALF_BLOCK_SIZE, HALF_BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim2.x, d_output);

  /* 8. Sequential Addressing (Tuning) 
      - Level 0: CEIL_DIV(33554432, 512) = 65536 blocks
      - Level 1: CEIL_DIV(65536, 512) = 128 blocks
      - Level 2: CEIL_DIV(128, 512) = 1 blocks
  */
  // const int DOUBLE_BLOCK_SIZE = BLOCK_SIZE * 2;
  // dim3 gridDim0(CEIL_DIV(size, DOUBLE_BLOCK_SIZE));
  // dim3 gridDim1(CEIL_DIV(gridDim0.x, DOUBLE_BLOCK_SIZE));
  // dim3 gridDim2(CEIL_DIV(gridDim1.x, DOUBLE_BLOCK_SIZE));
  // sequential_tuning_kernel<<<gridDim0, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_arr, size, d_output);
  // sequential_tuning_kernel<<<gridDim1, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim0.x, d_output);
  // sequential_tuning_kernel<<<gridDim2, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(d_output, gridDim1.x, d_output);
  
  /* 9. Full Warp-level Reduction 
      - Level 0: CEIL_DIV(33554432, 512) = 65536 blocks
      - Level 1: CEIL_DIV(65536, 512) = 128 blocks
      - Level 2: CEIL_DIV(128, 512) = 1 blocks
  */
  const int DOUBLE_BLOCK_SIZE = BLOCK_SIZE * 2;
  dim3 gridDim0(CEIL_DIV(size, DOUBLE_BLOCK_SIZE));
  dim3 gridDim1(CEIL_DIV(gridDim0.x, DOUBLE_BLOCK_SIZE));
  dim3 gridDim2(CEIL_DIV(gridDim1.x, DOUBLE_BLOCK_SIZE));
  full_warp_shfl_kernel<<<gridDim0, BLOCK_SIZE>>>(d_arr, size, d_output);
  full_warp_shfl_kernel<<<gridDim1, BLOCK_SIZE>>>(d_output, gridDim0.x, d_output);
  full_warp_shfl_kernel<<<gridDim2, BLOCK_SIZE>>>(d_output, gridDim1.x, d_output);

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void reduction_cublas(double* arr, int size) {
  const double alpha = 1.0, beta = 0.0;

  /* y [1 x 1] = A^T [1 x size] * x [size x 1] */
  CHECK_CUBLAS(hipblasDgemv(handle, 
    HIPBLAS_OP_T,        // To transpose
    size, 1,            // Matrix: [size x 1]
    &alpha,
    d_arr, size,        // lda = size
    d_ones, 1,          // Vector: [size x 1]
    &beta,
    d_cublas_output, 1));      // Output: [1 x 1]
  
  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void reduction_cublas_v2(double* arr, int size) {
  CHECK_CUBLAS(hipblasDdot(handle, size, d_arr, 1, d_ones, 1, d_cublas_output));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void reduction_initialize(double* arr, int size) {
  CHECK_CUDA(hipMalloc(&d_arr, size * sizeof(double)));
  CHECK_CUDA(hipMemcpy(d_arr, arr, size * sizeof(double), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&d_output, size * sizeof(double)));
}

void cublas_initialize(int size) {
  CHECK_CUBLAS(hipblasCreate(&handle));
  // CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH)); /* To enable TC */
  
  CHECK_CUDA(hipMalloc(&d_cublas_output, sizeof(double)));

  CHECK_CUDA(hipMalloc(&d_ones, size * sizeof(double)));
  double* h_ones = new double[size];
  for (int i = 0; i < size; i++) h_ones[i] = 1.0;
  hipMemcpy(d_ones, h_ones, size * sizeof(double), hipMemcpyHostToDevice);
  delete[] h_ones;
}

void reduction_finalize(double* output) {
  if (output != nullptr)
    CHECK_CUDA(hipMemcpy(output, d_output, sizeof(double), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(d_arr));
  CHECK_CUDA(hipFree(d_output));
}

void cublas_finalize(double* output) {
  if (output != nullptr)
    CHECK_CUDA(hipMemcpy(output, d_cublas_output, sizeof(double), hipMemcpyDeviceToHost));
  
  CHECK_CUDA(hipFree(d_cublas_output));
  CHECK_CUDA(hipFree(d_ones));

  CHECK_CUBLAS(hipblasDestroy(handle));
}